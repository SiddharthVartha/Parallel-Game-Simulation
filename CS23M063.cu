#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <chrono>

using namespace std;

//*******************************************

// Write down the kernels here

__device__ int c=0;
__constant__ int d_x[1000];
__constant__ int d_y[1000];

__global__ void begin(int round,int *d_health,int *d_score,int T){
    int start=threadIdx.x;
    int end=(start+round)%T;
    
    int x_start=d_x[start];
    int y_start=d_y[start];

    int x_end=d_x[end];
    int y_end=d_y[end];


    int distance=INT_MAX;
    
    if(d_health[start]>0){             
        if(d_health[end]>0){
          
          distance=abs(x_end - x_start) + abs(y_end - y_start);
        }
      
        for(int j=0;j<T;j++){
          int x_j=d_x[j],y_j=d_y[j];

          int temp=abs(x_j - x_start)+ abs(y_j - y_start);
          int condition=((x_end - x_start) * (y_j - y_start)) == ((y_end - y_start) * (x_j - x_start)) && d_health[j] > 0 && temp < distance && j!=start;
          if(condition){
            if (x_end>x_start && x_j>x_start) {
              distance=temp;
              end=j;
            }
            else if (x_end<x_start && x_j<x_start) {
                distance=temp;
                end=j;
            }
            else if (x_end==x_start && y_start<y_end && y_j>y_start) {
                distance=temp;
                end=j;
            }
            else if (x_end==x_start && y_start>y_end && y_j<y_start) {
                distance=temp;
                end=j;
            }
            
          }
          
          
      }

    }
    
    __syncthreads();
    
    
    if(distance!=INT_MAX){
        atomicAdd(&d_score[start], 1);
        atomicSub(&d_health[end], 1);
        
    }
    
    atomicAdd(&c, 1);
}

__global__ void playGame(int *d_health,int *d_score,int T){
    int round=1;
    
    while(1){
        if(round%T!=0){
            begin<<<1,T>>>(round,d_health,d_score,T);
        }
        else{
            atomicAdd(&c, T);
        }
        
        while (atomicCAS(&c, T, 0) < T);
        round+=1;
        int count=0;
        for(int i=0;i<T;i++){
            if(d_health[i]>0){
                count+=1;
            }
            if(count>1){
                break;
            }
        }
        if(count==0 || count==1){
            break;
        }
        
    }

}

__global__ void initializeArray(int *array, int value,int T) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    array[tid] = value;  
}

//***********************************************


int main(int argc,char **argv)
{
    // Variable declarations
    int M,N,T,H,*xcoord,*ycoord,*score;
    

    FILE *inputfilepointer;
    
    //File Opening for read
    char *inputfilename = argv[1];
    inputfilepointer    = fopen( inputfilename , "r");

    if ( inputfilepointer == NULL )  {
        printf( "input.txt file failed to open." );
        return 0; 
    }

    fscanf( inputfilepointer, "%d", &M );
    fscanf( inputfilepointer, "%d", &N );
    fscanf( inputfilepointer, "%d", &T ); // T is number of Tanks
    fscanf( inputfilepointer, "%d", &H ); // H is the starting Health point of each Tank
	
    // Allocate memory on CPU
    xcoord=(int*)malloc(T * sizeof (int));  // X coordinate of each tank
    ycoord=(int*)malloc(T * sizeof (int));  // Y coordinate of each tank
    score=(int*)malloc(T * sizeof (int));  // Score of each tank (ensure that at the end you have copied back the score calculations on the GPU back to this allocation)

    // Get the Input of Tank coordinates
    for(int i=0;i<T;i++)
    {
      fscanf( inputfilepointer, "%d", &xcoord[i] );
      fscanf( inputfilepointer, "%d", &ycoord[i] );
    }
		

    auto start = chrono::high_resolution_clock::now();

    //*********************************
    // Your Code begins here (Do not change anything in main() above this comment)
    //********************************
    
    int *d_health,*d_score;

    hipMalloc(&d_health,sizeof(int)*T);
    initializeArray<<<1, T>>>(d_health, H,T);
    hipDeviceSynchronize();
    hipMalloc(&d_score,sizeof(int)*T);
    initializeArray<<<1, T>>>(d_score, 0,T);
    hipDeviceSynchronize();
    hipMemcpyToSymbol(HIP_SYMBOL(d_x), xcoord, sizeof(int)*T);
    hipMemcpyToSymbol(HIP_SYMBOL(d_y), ycoord, sizeof(int)*T);

    playGame<<<1,1>>>(d_health,d_score,T);
    hipMemcpy(score,d_score,sizeof(int)*T,hipMemcpyDeviceToHost);
    //*********************************
    // Your Code ends here (Do not change anything in main() below this comment)
    //********************************

    auto end  = chrono::high_resolution_clock::now();

    chrono::duration<double, std::micro> timeTaken = end-start;

    printf("Execution time : %f\n", timeTaken.count());

    // Output
    char *outputfilename = argv[2];
    char *exectimefilename = argv[3]; 
    FILE *outputfilepointer;
    outputfilepointer = fopen(outputfilename,"w");

    for(int i=0;i<T;i++)
    {
        fprintf( outputfilepointer, "%d\n", score[i]);
    }
    fclose(inputfilepointer);
    fclose(outputfilepointer);

    outputfilepointer = fopen(exectimefilename,"w");
    fprintf(outputfilepointer,"%f", timeTaken.count());
    fclose(outputfilepointer);

    free(xcoord);
    free(ycoord);
    free(score);
    hipDeviceSynchronize();
    return 0;
}